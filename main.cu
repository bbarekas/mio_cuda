
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void cuda_add(int n, float *x, float *y)
{

    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}


// function to add the elements of two arrays
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int native_main(void)
{
    int N = 1<<20; // 1M elements

    float *x = new float[N];
    float *y = new float[N];

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    auto start = std::chrono::high_resolution_clock::now();

    // Run kernel on 1M elements on the CPU
    add(N, x, y);

    auto finish = std::chrono::high_resolution_clock::now();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;
    std::cout << "T: " << std::chrono::duration_cast<std::chrono::microseconds>(finish-start).count() << "ns\n";

    // Free memory
    delete [] x;
    delete [] y;

    return 0;
}

int cuda_main(void)
{
    int N = 1<<20; // 1M elements

    float *x;
    float *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    auto start = std::chrono::high_resolution_clock::now();

    // Run kernel on 1M elements on the CPU
    //cuda_add<<<1, 1>>>(N, x, y);
    cuda_add<<<1, 1024>>>(N, x, y);

    auto finish = std::chrono::high_resolution_clock::now();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "CUDA - Max error: " << maxError << std::endl;
    std::cout << "T: " << std::chrono::duration_cast<std::chrono::microseconds>(finish-start).count() << "ns\n";

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

#if 0

static void bench_singleT(benchmark::State &state) {
    for (auto _ : state)
        single_thread();
}
//BENCHMARK(bench_singleT);
BENCHMARK(bench_singleT)->UseRealTime()->Unit(benchmark::kMillisecond);

static void bench_multiT_false(benchmark::State &state) {
    for (auto _ : state)
        multi_thread_false();
}
//BENCHMARK(bench_multiT);
BENCHMARK(bench_multiT_false)->UseRealTime()->Unit(benchmark::kMillisecond);


// Run the benchmark
BENCHMARK_MAIN();

#else

//
int main() {
    std::cout << "Hello, World!" << std::endl;

    // Native version
    native_main();

    // CUDA version
    cuda_main();

    return 0;
}

#endif